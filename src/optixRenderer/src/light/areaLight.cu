#include "hip/hip_runtime.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "lightStructs.h"
#include "light/areaLight.h"
#include "structs/prd.h"
#include "random.h"
#include "commonStructs.h"


using namespace optix;

rtDeclareVariable( float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable( float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable( float, t_hit, rtIntersectionDistance, ); // with this the hitPoint can be calculated
rtDeclareVariable(int, max_depth, , );


rtDeclareVariable(optix::Ray, ray,   rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow, rtPayload, );

rtDeclareVariable(float3, radiance, , );
rtDeclareVariable(float, areaSum, , );

rtDeclareVariable(int, areaTriangleNum, , );
rtBuffer<areaLight> areaLights;
rtBuffer<float> areaLightCDF;
rtBuffer<float> areaLightPDF;


RT_CALLABLE_PROGRAM void sampleAreaLight(unsigned int& seed, float3& radiance, float3& position, float3& normal, float& pdfAreaLight){
    float randf = rnd(seed);

    int left = 0, right = areaTriangleNum;
    int middle = int( (left + right) / 2);
    while(left < right){
        if(areaLightCDF[middle] <= randf)
            left = middle + 1;
        else if(areaLightCDF[middle] > randf)
            right = middle;
        middle = int( (left + right) / 2);
    }
    areaLight L = areaLights[left];
    
    float3 v1 = L.vertices[0];
    float3 v2 = L.vertices[1];
    float3 v3 = L.vertices[2];

    normal = cross(v2 - v1, v3 - v1);
    float area = 0.5 * length(normal);
    normal = normalize(normal);

    float ep1 = rnd(seed);
    float ep2 = rnd(seed);
    
    float u = 1 - sqrt(ep1);
    float v = ep2 * sqrt(ep1);

    position = v1 + (v2 - v1) * u + (v3 - v1) * v;

    radiance = L.radiance;
    pdfAreaLight = areaLightPDF[left] /  fmaxf(area, 1e-14);
}

RT_PROGRAM void closest_hit_radiance()
{
    const float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    const float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    const float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

    if(prd_radiance.depth == 0){
        // Directly hit the light
        prd_radiance.radiance = radiance;
    }
    else{
        if(prd_radiance.pdf < 0){
            prd_radiance.radiance += radiance * prd_radiance.attenuation;
        }
        else{
            // Use MIS to compute the radiance
            if(prd_radiance.depth == (max_depth - 1) ){
                prd_radiance.radiance += radiance * prd_radiance.attenuation;
            }
            else{
                float3 hitPoint = ray.origin + t_hit * ray.direction;
                float Dist = length(hitPoint - prd_radiance.origin);
                float3 L = normalize(hitPoint - prd_radiance.origin);
                float cosPhi = dot(L, ffnormal);
                if (cosPhi < 0) cosPhi = -cosPhi;
                if (cosPhi < 1e-14) cosPhi = 0;
        
                float pdfAreaBRDF = prd_radiance.pdf * cosPhi / Dist / Dist;
                float pdfAreaLight = length(radiance) / areaSum;

                float pdfAreaBRDF2 = pdfAreaBRDF * pdfAreaBRDF;
                float pdfAreaLight2 = pdfAreaLight * pdfAreaLight;
       
                prd_radiance.radiance += radiance * pdfAreaBRDF2 / fmaxf(pdfAreaBRDF2 + pdfAreaLight2, 1e-14) * prd_radiance.attenuation;
            }
        }
    }
    prd_radiance.done = true;
}


RT_PROGRAM void any_hit_shadow()
{
    prd_shadow.inShadow = true;
    rtTerminateRay();
}
