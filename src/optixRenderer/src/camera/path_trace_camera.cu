#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "structs/prd.h"
#include "random.h"

using namespace optix;


rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        cameraU, , );
rtDeclareVariable(float3,        cameraV, , );
rtDeclareVariable(float3,        cameraW, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(int,           max_depth, , );
rtBuffer<float3, 2>              output_buffer;
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(uint2,         launch_index, rtLaunchIndex, );
rtDeclareVariable(unsigned int, sqrt_num_samples, , );
rtDeclareVariable(unsigned int, rr_begin_depth, , );
rtDeclareVariable(int, cameraMode, , );
rtDeclareVariable(unsigned int, initSeed, , );

RT_PROGRAM void pinhole_camera()
{
    size_t2 screen = output_buffer.size();
    
    float2 inv_screen = 1.0f/make_float2(screen) * 2.f;
    float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;

    float2 jitter_scale = inv_screen / sqrt_num_samples;
    unsigned int samples_per_pixel = sqrt_num_samples*sqrt_num_samples;
    float3 radiance = make_float3(0.0f);

    unsigned int seed = tea<32>( 
        ( (initSeed)*(screen.x*launch_index.y+launch_index.x) + initSeed ), 
        ( (screen.y * launch_index.x + launch_index.y) * initSeed ) );
    
    do{
        // Sample pixel using jittering
        float3 ray_origin = eye;
        unsigned int x = samples_per_pixel%sqrt_num_samples;
        unsigned int y = samples_per_pixel/sqrt_num_samples;
        float2 jitter = make_float2(x-rnd(seed), y-rnd(seed) );
        float2 d = pixel + jitter*jitter_scale;

        
        float3 ray_direction;
        if(cameraMode == 0){
            ray_direction = normalize(d.x*cameraU + d.y*cameraV + cameraW);
        }
        else{
            float3 axisZ = normalize(cameraW );
            float3 axisX = normalize(cameraU );
            float3 axisY = normalize(cameraV );

            d.x = (d.x > 1.0f) ? 1.0f : d.x;
            d.y = (d.y > 1.0f) ? 1.0f : d.y;
            d.x = (d.x < -1.0f) ? -1.0f : d.x;
            d.y = (d.y < -1.0f) ? -1.0f : d.y;

            float phi = d.x * M_PIf;
            float theta;
            if(cameraMode == 1){
                theta = 0.5f * (-d.y + 1.0f) * M_PIf;
            }
            else if(cameraMode == 2){
                theta = 0.25f * (-d.y + 1.0f) * M_PIf;
            }
            ray_direction = normalize(
                    sinf(theta) * cosf(phi) * axisX 
                    + sinf(theta) * sinf(phi) * axisY 
                    + cosf(theta) * axisZ
                    );
        }

        // Initialze per-ray data
        PerRayData_radiance prd;
        prd.attenuation = make_float3(1.f);
        prd.radiance = make_float3(0.f);
        prd.done = false;
        prd.seed = seed;
        prd.depth = 0;
        prd.direction = ray_direction;

        // Each iteration is a segment of the ray path.  The closest hit will
        // return new segments to be traced here.
        for(;;)
        {
            Ray ray(ray_origin, ray_direction, 0, scene_epsilon);
            rtTrace(top_object, ray, prd);
 
            prd.depth++; 
           
            if(prd.depth > rr_begin_depth){
                float z = rnd(prd.seed);
                if(z < length(prd.attenuation) ){
                    prd.attenuation = prd.attenuation / fmaxf(length(prd.attenuation), 1e-10);
                }
                else{
                    prd.done = true;
                }
            }

            // Hit the light source or exceed the max depth
            if(prd.done || prd.depth >= max_depth) 
                break; 
            
            // Update ray data for the next path segment
            ray_origin = prd.origin;
            ray_direction = prd.direction;
        }
        radiance += prd.radiance;
        seed = prd.seed;
    } while (--samples_per_pixel);

    // Update the output buffer
    float3 pixel_color = radiance / (sqrt_num_samples*sqrt_num_samples);
    output_buffer[launch_index] = pixel_color;
}

RT_PROGRAM void exception()
{
  const unsigned int code = rtGetExceptionCode();
  rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
  output_buffer[launch_index] = bad_color;
}
