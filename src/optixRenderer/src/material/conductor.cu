#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "structs/prd.h"
#include "random.h"
#include "commonStructs.h"
#include "lightStructs.h"
#include <vector> 

using namespace optix; 

rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, tangent_direction, attribute tangent_direction, );
rtDeclareVariable(float3, bitangent_direction, attribute bitangent_direction, );

rtDeclareVariable( float3, texcoord, attribute texcoord, );
rtDeclareVariable( float, t_hit, rtIntersectionDistance, );

rtDeclareVariable(optix::Ray, ray,   rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow, rtPayload, );
rtDeclareVariable(float, scene_epsilon, , );

// Normal 
rtTextureSampler<float4, 2> normalMap;
rtDeclareVariable( int, isNormalTexture, , );

// Material 
rtDeclareVariable(float3, specular, , );

// Geometry Group
rtDeclareVariable( rtObject, top_object, , );

// Computing the pdfSolidAngle of BRDF giving a direction 
RT_CALLABLE_PROGRAM float pdf(const float3& L, const float3& V, const float3& N)
{
    return 0.0f;
}

RT_CALLABLE_PROGRAM float3 evaluate(const float3& N, const float3& V, const float3& L, const float3& radiance)
{   
    return make_float3(0.0f);
}
 
RT_CALLABLE_PROGRAM void sample(unsigned& seed, 
        float3 N, const float3& V, 
        float3& attenuation, float3& direction, float& pdfSolid)
{
    pdfSolid = -1;
    direction = reflect(-V, N);   
    attenuation = attenuation * specular;
}


RT_PROGRAM void closest_hit_radiance()
{
    const float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    const float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

    float3 V = normalize(-ray.direction );
    
    float3 N;
    if( isNormalTexture == 0){
        N = world_shading_normal;
    }
    else{
        N = make_float3(tex2D(normalMap, texcoord.x, texcoord.y) );
        N = normalize(2 * N - 1);
        N = N.x * tangent_direction 
            + N.y * bitangent_direction 
            + N.z * world_shading_normal;
    }
    N = normalize(N );
    
    float3 hitPoint = ray.origin + t_hit * ray.direction;
    prd_radiance.origin = hitPoint;


    // Finish updating the ray
    sample(prd_radiance.seed,
            N, V,
            prd_radiance.attenuation, prd_radiance.direction, prd_radiance.pdf 
            );
}

// any_hit_shadow program for every material include the lighting should be the same
RT_PROGRAM void any_hit_shadow()
{
    prd_shadow.inShadow = true;
    rtTerminateRay();
}

