#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "structs/prd.h"
#include "random.h"
#include "commonStructs.h"
#include "lightStructs.h"
#include <vector>

using namespace optix;

rtDeclareVariable( float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable( float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, tangent_direction, attribute tangent_direction, );
rtDeclareVariable(float3, bitangent_direction, attribute bitangent_direction, );
rtDeclareVariable( float3, texcoord, attribute texcoord, );
rtDeclareVariable(optix::Ray, ray,   rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow, rtPayload, );

rtDeclareVariable( float, uvScale, , ); 
rtTextureSampler<float4, 2> normalMap;
rtDeclareVariable(int, isNormalTexture, , );
rtDeclareVariable(float, F0, , );

rtDeclareVariable( float3, cameraU, , );
rtDeclareVariable( float3, cameraV, , );
rtDeclareVariable( float3, cameraW, , );

RT_PROGRAM void closest_hit_radiance()
{
    const float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    const float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );
    
    float3 V = normalize(-ray.direction );    
    
    if(dot(ffnormal, V) < 0)
        ffnormal = -ffnormal;

    float3 N;
    if( isNormalTexture == 0){
        N = ffnormal;
    }
    else{
        N = make_float3(tex2D(normalMap, texcoord.x * uvScale, texcoord.y * uvScale) );
        N = normalize(2 * N - 1);
        N = N.x * tangent_direction 
            + N.y * bitangent_direction 
            + N.z * ffnormal; 
    }
    N = normalize(N );


    float3 Z = normalize(-cameraW);
    float3 X = normalize(cameraU);
    float3 Y = normalize(cameraV);
    N = make_float3( dot(N, X), dot(N, Y), dot(N, Z) );

    prd_radiance.radiance = 0.5 * (N + 1);
    prd_radiance.done = true;
}

// any_hit_shadow program for every material include the lighting should be the same
RT_PROGRAM void any_hit_shadow()
{
    prd_shadow.inShadow = true;
    rtTerminateRay();
}

